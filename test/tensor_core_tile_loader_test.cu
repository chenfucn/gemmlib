#include "hip/hip_runtime.h"
/**
 * Copyright (c) Microsoft.
 * Licensed under the MIT license.
 *
 * @file tensor_core_tile_loader_test.cu
 */

#include <hip/hip_runtime.h>
#include "cutlass/aligned_buffer.h"
#include "cutlass/cutlass.h"
#include "cutlass/device_kernel.h"
#include "cutlass/gemm_coord.h"
#include "cutlass/matrix_shape.h"

#include "cutlass/arch/mma.h"
#include "cutlass/gemm/warp/mma_tensor_op.h"
#include "cutlass/gemm/warp/mma_tensor_op_policy.h"

#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/debug.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "matrix_layout.h"
#include "blkq4_fp16_util.h"
#include "blkq4_fp16_gemm_sm80.h"
#include "ref_gemm.h"

#include "gemm/warp/tensor_core_tile_loader.h"
#include "gemm/warp/quantb_meta_loader.h"

#include "gtest/gtest.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace onnxruntime {
namespace cuda {
namespace test {

/////////////////////////////////////////////////////////////////////////////////////////////////

template <
  typename QuantBlocking_,              ///! Shape of the quantization block, either 1xb or bx1
  bool     has_quant_offset_,           ///! Whether the quantization has offset
  typename WarpShape_,                  ///! Warp-scoped matrix multiply-accumulate
  int SplitKSerial_ = 1,                ///! How many warps to split the K dimension in the same MxN block
  int Stages_ = 4                       ///! Stages of the pipelined mainloop
>
struct LoadPackedBTestKernel {
 public:
  //
  // Type definitions
  //

  using QuantBlocking = QuantBlocking_;
  using WarpShape = WarpShape_;
  static constexpr bool has_quant_offset = has_quant_offset_;
  static constexpr int kSplitK = SplitKSerial_;
  static constexpr int kStages = Stages_;

  static_assert(kSplitK > 0 && ((kSplitK - 1) & kSplitK) == 0,
     "SplitK must be positive and a power of 2");
  static_assert(kStages > 1,
     "Number of pipeline stages must be greater than 1.");

  /// switches for debug print
  static constexpr bool kDebugPrintB = false;
  static constexpr bool kDebugPrintFragA = false;
  static constexpr bool kDebugPrintC = false;

  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
  using ElementT = cutlass::half_t;
  static constexpr int kElementSize = sizeof(ElementT);
  static_assert(kElementSize == 2, "Only support 16b float now");

  // Quantized weights are packed int4, each 16x16 tile of int4
  // is packed into 8x8 tile of 16b (i.e. 8x16 tile of bytes)
  static_assert(WarpShape::kN % 16 == 0 && WarpShape::kK % 16 == 0,
    "Weight B is packed as 16x16 tiles, warp shape must contain whole tiles!");
  using WarpPackedBShape = cutlass::gemm::GemmShape<1, WarpShape::kN/2, WarpShape::kK>;

  // decide per warp tile loader shape, it loads 1, 2 or 4 tiles at a time
  static constexpr int kNTilesPerLoad = std::min(4, WarpPackedBShape::kN / 8);
  static constexpr int kKTilesPerLoad = std::min(4/kNTilesPerLoad, WarpPackedBShape::kK / 16);
  using PackedBLoader = mickey::gemm::warp::TensorCoreTileLoader<kNTilesPerLoad, kKTilesPerLoad>;

  static_assert((WarpPackedBShape::kN % PackedBLoader::kMNStride) == 0);
  static_assert((WarpPackedBShape::kK % PackedBLoader::kKStride) == 0);

  static constexpr int kB_Nloads = WarpPackedBShape::kN / PackedBLoader::kMNStride;
  static constexpr int kB_Kloads = WarpPackedBShape::kK / PackedBLoader::kKStride;

  using MetaLoader = mickey::gemm::warp::QuantBScaleLoader<QuantBlocking, WarpShape, ElementT, false>;

  // Need to explore the way to relax this for very small m value.
  static_assert(WarpShape::kM % 16 == 0,
      "Stride M smaller than mma instruction shape is not yet supported!");

  // load A to shared memory, 2x2 tile to match the tensorcore shape 16x8x16
  using ATileLoader = mickey::gemm::warp::TensorCoreTileLoader<2, 2>;
  static_assert(ATileLoader::kMNStride == InstructionShape::kM);
  static_assert(ATileLoader::kKStride == InstructionShape::kK * kElementSize);
  static constexpr int kA_Mloads = WarpShape::kM / InstructionShape::kM;
  static constexpr int kA_Kloads = WarpShape::kK / InstructionShape::kK;

  // Since int4 weights are packed (16x16) -> (8x8), each tile is expanded to 4 tiles when
  // de-quantized to 16b float.

  // Fragments of quantized weights, keep all warp tile in registers for it's easier to
  // locate corresponding scale
  using FragmentPackedB = cutlass::Array<
      unsigned,  // 8 of int4 weights each tile (becomes 4 tiles when de-quantized)
      PackedBLoader::kTiles * kB_Nloads * kB_Kloads>;

  // Fragments for operand A and B, each tile has 2 elements per thread. In each iteration,
  // we use a (WarpShape::kM, 16) block of A and (16, WarpShape::kN) block of B for mma
  using FragmentA = cutlass::Array<ElementT, 2 * (WarpShape::kM / 8) * 2>;
  using FragmentB = cutlass::Array<ElementT, 2 * (WarpShape::kN / 8) * 2>;

  static constexpr int kWarps = kSplitK; // TODO! more warps when we have a larger thread block shape
  static int const kThreadCount = 32 * kWarps;

  //
  // The way we use the cutlass MmaTensorOp class below is confusing, because:
  //
  // MmaTensorOp from cutlass is really convoluted. It iterates over the m,n
  // dimension to run mma instructions the following number of times:
  // (WarpShape::kM / InstructionShape::kM) * (WarpShape::kN / InstructionShape::kN).
  // So, the operation always cover a shape of
  // (WarpShape::kM, WarpShape::kN, InstructionShape::kK).
  // Unfortunately, it does not reach that conclusion in a straight forward
  // way. Instead, it asks you to provide a shared memory layout for both A
  // and B, and construct shared memory tile iterators based on these layout.
  // The solo purpose of these iterators is to compute the k dimension size.
  // And they don't access shared memory at all. What's worse, the layout
  // must be a certain swizzled shape, for it to compute the current k, or
  // else the operation can not be used. This is a serious abstraction leak
  // that makes this class difficult to use.
  //

  using MmaPolicy = cutlass::gemm::warp::MmaTensorOpPolicy<
      cutlass::arch::Mma<InstructionShape, 32, ElementT,
                         cutlass::layout::RowMajor, ElementT,
                         cutlass::layout::ColumnMajor, float,
                         cutlass::layout::RowMajor, cutlass::arch::OpMultiplyAdd>,
      cutlass::MatrixShape<1, 1> >;

  using MmaOp = cutlass::gemm::warp::MmaTensorOp<
      cutlass::gemm::GemmShape<WarpShape::kM, WarpShape::kN, InstructionShape::kK>, ElementT, cutlass::layout::RowMajor, ElementT,
      cutlass::layout::ColumnMajor, float, cutlass::layout::RowMajor,
      MmaPolicy>;

  /// Parameters structure
  struct Params {
    cutlass::gemm::GemmCoord problem_size_;

    // Decide thread block level partitioning. Here the K value is always 1,
    // as we don't split K dimension at thread block level. Instead, we split
    // K dimension at warp level based on template parameter SplitKSerial_.
    cutlass::gemm::GemmCoord grid_tiled_shape_;
    void* const ptr_output_;
    const int output_byte_stride_;
    void const * const ptr_a_;
    const int a_byte_stride_;
    void const * const ptr_packed_b_;
    const int b_byte_stride_;
    void const * const ptr_scales_;
    const int scales_byte_stride_;
    void const * const ptr_offsets_;
    const int offsets_byte_stride_;
    int gemm_k_size_{0};

    CUTLASS_HOST_DEVICE
    Params() { }

    CUTLASS_HOST_DEVICE
    Params(
      cutlass::gemm::GemmCoord const & problem_size,
      void* ptr_output,
      int output_byte_stride,
      void const *ptr_a,
      int a_byte_stride,
      void const *ptr_packed_b,
      int b_byte_stride,
      void const *ptr_scales,
      int scales_byte_stride,
      void const *ptr_offsets = nullptr,
      int offsets_byte_stride = 0
    ):
      problem_size_(problem_size),
      ptr_output_(ptr_output),
      output_byte_stride_(output_byte_stride),
      ptr_a_(ptr_a),
      a_byte_stride_(a_byte_stride),
      ptr_packed_b_(ptr_packed_b),
      b_byte_stride_(b_byte_stride),
      ptr_scales_(ptr_scales),
      scales_byte_stride_(scales_byte_stride),
      ptr_offsets_(ptr_offsets),
      offsets_byte_stride_(offsets_byte_stride),
      gemm_k_size_(mickey::round_up(mickey::div_up(problem_size.k(), kSplitK), WarpShape::kK)),
      // TODO! grid_tiled_shape_ should be based on thread block shape
      grid_tiled_shape_(cutlass::gemm::GemmCoord(
        mickey::div_up(problem_size.m(), WarpShape::kM),
        mickey::div_up(problem_size.n(), WarpShape::kN),
        1)) { }
  };

  /// Shared memory storage structure
  struct SharedStorage {
    /// Buffer for prepacked weights
    static constexpr int kPackedBSizePerIter = kB_Nloads * kB_Kloads * PackedBLoader::kByteSize;
    static constexpr int kPackedBSizePerWarp = kPackedBSizePerIter * kStages;
    static constexpr int kPackedBSize = kPackedBSizePerWarp * kWarps;
    cutlass::AlignedBuffer<uint8_t, kPackedBSize> shared_B;

    /// Buffer for A tensor
    static constexpr int kASizePerIter = kA_Mloads * kA_Kloads * ATileLoader::kByteSize / kElementSize;
    static constexpr int kASizePerWarp = kASizePerIter * kStages;
    static constexpr int kASize = kASizePerWarp * kWarps;
    cutlass::AlignedBuffer<ElementT, kASize> shared_A;

    /// Buffer for accumulators of the partial results
    static constexpr int kAccSizePerWarp = WarpShape::kM * WarpShape::kN;
    static constexpr int kAccSize = kAccSizePerWarp * (kWarps - 1);
    cutlass::AlignedBuffer<float, kAccSize> shared_Acc;
  };

  //
  // Methods
  //

  CUTLASS_HOST_DEVICE
  LoadPackedBTestKernel() { }

  /// Determines whether kernel satisfies alignment
  static cutlass::Status can_implement(const Params &params) {
    if ((reinterpret_cast<uintptr_t>(params.ptr_a_) % 16)) {
      std::cerr << "LoadPackedBTestKernel validation fail: params.ptr_a_ is not aligned to 16 bytes!" << std::endl;
      return cutlass::Status::kErrorMisalignedOperand;
    }
    if (params.a_byte_stride_ % 16) {
      std::cerr << "LoadPackedBTestKernel validation fail: params.a_byte_stride_ is not aligned to 16 bytes!" << std::endl;
      return cutlass::Status::kErrorMisalignedOperand;
    }
    if ((params.problem_size_.k() % QuantBlocking::kRow != 0) ||
        (params.problem_size_.n() % QuantBlocking::kColumn) != 0){
      std::cerr << "LoadPackedBTestKernel validation fail: partial quantization block not supported!" << std::endl;
      return cutlass::Status::kErrorInvalidProblem;
    }
    if (reinterpret_cast<uintptr_t>(params.ptr_packed_b_) % 16) {
      std::cerr << "LoadPackedBTestKernel validation fail: params.ptr_packed_b_ is not aligned to 16 bytes!" << std::endl;
      return cutlass::Status::kErrorMisalignedOperand;
    }
    if (params.b_byte_stride_ % 16) {
      std::cerr << "LoadPackedBTestKernel validation fail: params.b_byte_stride_ is not aligned to 16 bytes!" << std::endl;
      return cutlass::Status::kErrorMisalignedOperand;
    }
    if (reinterpret_cast<uintptr_t>(params.ptr_scales_) % 16) {
      std::cerr << "LoadPackedBTestKernel validation fail: params.ptr_scales_ is not aligned to 16 bytes!" << std::endl;
      return cutlass::Status::kErrorMisalignedOperand;
    }
    if (params.scales_byte_stride_ % 16) {
      std::cerr << "LoadPackedBTestKernel validation fail: params.scales_byte_stride_ is not aligned to 16 bytes!" << std::endl;
      return cutlass::Status::kErrorMisalignedOperand;
    }
    if constexpr (has_quant_offset) {
      if (params.ptr_offsets_ == nullptr || params.offsets_byte_stride_ == 0) {
        std::cerr << "LoadPackedBTestKernel validation fail: Required quantization offsets are not provided!" << std::endl;
        return cutlass::Status::kErrorInvalidProblem;
      }
      if (reinterpret_cast<uintptr_t>(params.ptr_offsets_) % 16) {
        std::cerr << "LoadPackedBTestKernel validation fail: params.ptr_offsets_ is not aligned to 16 bytes!" << std::endl;
        return cutlass::Status::kErrorMisalignedOperand;
      }
      if (params.offsets_byte_stride_ % 16) {
        std::cerr << "LoadPackedBTestKernel validation fail: params.offsets_byte_stride_ is not aligned to 16 bytes!" << std::endl;
        return cutlass::Status::kErrorMisalignedOperand;
      }
    } else {
      if (params.ptr_offsets_ != nullptr || params.offsets_byte_stride_ != 0) {
        std::cerr << "LoadPackedBTestKernel validation fail: quantization offsets are provided to scale only kernel!" << std::endl;
        return cutlass::Status::kErrorInvalidProblem;
      }
    }

    if (reinterpret_cast<uintptr_t>(params.ptr_output_) % 16) {
      std::cerr << "LoadPackedBTestKernel validation fail: params.ptr_output_ is not aligned to 16 bytes!" << std::endl;
      return cutlass::Status::kErrorMisalignedOperand;
    }
    if (params.output_byte_stride_ % 16) {
      std::cerr << "LoadPackedBTestKernel validation fail: params.output_byte_stride_ is not aligned to 16 bytes!" << std::endl;
      return cutlass::Status::kErrorMisalignedOperand;
    }
    if (params.problem_size_.n() > (params.output_byte_stride_ / kElementSize)) {
      std::cerr << "LoadPackedBTestKernel validation fail: params.problem_size_.n() is greater than params.output_byte_stride_!" << std::endl;
      return cutlass::Status::kErrorInvalidProblem;
    }
    if (params.problem_size_.k() % 16 != 0) {
      std::cerr << "LoadPackedBTestKernel validation fail: params.problem_size_.k() is not aligned to 16 bytes!" << std::endl;
      return cutlass::Status::kErrorInvalidProblem;
    }
    if (params.problem_size_.k() > params.b_byte_stride_) {
      std::cerr << "LoadPackedBTestKernel validation fail: params.problem_size_.k() is greater than params.b_byte_stride_!" << std::endl;
      // for gemm of 16b floats, weights is packed to shape (k/2,n/2), column major
      // so stride should be greater or equal to k/2, with element size 2, it should be k
      return cutlass::Status::kErrorInvalidProblem;
    }

    if constexpr (kSplitK > 1){
      // TODO! Use thread block shape
      int remain = params.problem_size_.k() % params.gemm_k_size_;
      if (remain > 0 && remain < WarpShape::kK * kStages * 2) {
        // spliting too small, may not get enough iterations to rampup pipeline
        std::cerr << "LoadPackedBTestKernel validation fail: kSplitK is too small, k: " << remain << " is smaller than " << (WarpShape::kK * kStages * 4) << std::endl;
        return cutlass::Status::kErrorNotSupported;
      }
    }

    return cutlass::Status::kSuccess;
  }

  /// Executes one GEMM
  CUTLASS_DEVICE
  void operator()(Params const &params, SharedStorage &shared_storage) {
    // Early exit if CTA is out of range
    if (params.grid_tiled_shape_.m() <= blockIdx.x ||
      params.grid_tiled_shape_.n() <= blockIdx.y) {
      // should not happen
      if (threadIdx.x == 0) {
        printf("CTA out of range %d, %d\n", blockIdx.x, blockIdx.y);
      }
      return;
    }

    //
    // Initialization phase: locating our position
    //
    const int warp_idx = threadIdx.x / 32;
    const int lane_idx = threadIdx.x % 32;
    const int warp_idx_k = warp_idx % kSplitK;
    const int lane_b_k_offset = lane_idx % 4;
    const int lane_b_n_offset = lane_idx / 4;
#ifndef NDEBUG
    bool assert_pass = true;
    if (warp_idx >= kWarps) {
      assert_pass = false;
      if (lane_idx == 0) {
        printf("warp_idx %d exceeds kWarps %d! Should use %d threads per threadblock for kernel launch!\n",
          warp_idx, kWarps, kThreadCount);
      }
    }
    if (warp_idx_k != warp_idx) {
      assert_pass = false;
      if (lane_idx == 0) {
        printf("warp_idx_k %d should be equal to warp_idx %d while we don't yet specify thread block shape larger than warp shape!\n",
          warp_idx_k, warp_idx);
      }
    }
    assert(assert_pass);
#endif

    // Local fragments hopefully allocated in registers
    FragmentPackedB fragment_packed_b;
    typename MetaLoader::FragmentScales fragment_scales[kStages];
    FragmentB fragment_b;
    FragmentA fragment_a;
  
    typename MmaOp::FragmentC accumulators;
    accumulators.clear();
  
    MmaOp mma_op;

    //
    // for gemm input B size (k,n), packed b is (k/2,n/2), element size 2, column major.
    // so lead dimension byte size is coincidentally k/2 * 2 = k
    // and next dimension size is n/2
    //
    const int n_start = blockIdx.y * WarpShape::kN;   // TODO! change to thread block shape
    const int n_end = min(params.problem_size_.n(), (blockIdx.y + 1) * WarpShape::kN);
    const int packed_n_start = (n_start) / 2;
    const int packed_n_end = n_end / 2;
  
    const int k_start = warp_idx_k * params.gemm_k_size_;
    const int k_end = min(params.problem_size_.k(), (warp_idx_k + 1) * params.gemm_k_size_);

    const int m_start = blockIdx.x * WarpShape::kM;  // TODO! change to thread block shape
    const int m_end = min(params.problem_size_.m(), (blockIdx.x + 1) * WarpShape::kM);

    PackedBLoader packed_b_loader{
      params.ptr_packed_b_,
      params.b_byte_stride_,
      packed_n_start,
      packed_n_end,
      k_start,
      k_end,
      lane_idx};

    MetaLoader meta_loader{
      lane_idx,
      params.ptr_scales_,
      params.scales_byte_stride_,
      n_start, n_end};

    ATileLoader a_tile_loader{
      params.ptr_a_,
      params.a_byte_stride_,
      m_start, m_end,
      k_start * kElementSize, k_end * kElementSize, // need to convert to byte based index
      lane_idx};

    if constexpr (kDebugPrintB || kDebugPrintFragA || kDebugPrintC) {
      if (lane_idx == 0) {
        printf("Warp: %d, m_start %d, m_end %d, n_start %d, n_end %d, k_start %d, k_end %d, packed_n_start %d, packed_n_end %d\n",
          warp_idx, m_start, m_end, n_start, n_end, k_start, k_end, packed_n_start, packed_n_end);
      }
    }

    int load_k = k_start; // current k index for loading from global memory to shared memory
    int proc_k = k_start; // current k index for reading from shared memory and processing
    int smem_write_stage = 0;
    int smem_read_stage = 0;
    uint8_t* packed_b_shared_ptr = shared_storage.shared_B.data() + 
      SharedStorage::kPackedBSizePerWarp * warp_idx;
    ElementT* a_shared_ptr = shared_storage.shared_A.data() + 
      SharedStorage::kASizePerWarp * warp_idx;

    //
    // Prologue
    //
    CUTLASS_PRAGMA_UNROLL
    for (; smem_write_stage < kStages - 1; ++smem_write_stage, load_k += WarpShape::kK) {
      uint8_t* packed_b_smem_ptr = packed_b_shared_ptr + smem_write_stage * SharedStorage::kPackedBSizePerIter;
      ElementT* a_smem_ptr = a_shared_ptr + smem_write_stage * SharedStorage::kASizePerIter;

      meta_loader.load(fragment_scales[smem_write_stage], load_k, min(k_end, load_k + WarpShape::kK));

      // Load packed b
      CUTLASS_PRAGMA_UNROLL
      for (int k_load = 0; k_load < kB_Kloads; ++k_load) {
        packed_b_loader.load_lateral_n<kB_Nloads>(packed_b_smem_ptr);
        packed_b_smem_ptr += PackedBLoader::kByteSize * kB_Nloads;
        ++packed_b_loader;
      }

      // Load A
      CUTLASS_PRAGMA_UNROLL
      for (int ka_load = 0; ka_load < kA_Kloads; ++ka_load) {
        a_tile_loader.load_lateral_n<kA_Mloads>(a_smem_ptr);
        a_smem_ptr += ATileLoader::kByteSize * kA_Mloads / kElementSize;
        ++a_tile_loader;
      }

      // Defines the boundary of a stage of cp.async.
      cutlass::arch::cp_async_fence();
    }    

    // Wait until we have at least one committed global fetch stage. (#uncommitted = Base::kStages - 1 - #committed)
    cutlass::arch::cp_async_wait<kStages - 2>();
    //__syncthreads(); is this necessary since the loader is warp based?
    if constexpr(kDebugPrintB) {
      if (lane_idx == 0) {
        printf("Prologue, warp: %d, ShapredPtr: %p, WarpPtr: %p\n",
          warp_idx, shared_storage.shared_A.data(), a_shared_ptr);
        printf("\n********Dumping the shared memory of Warp %d*******\n\n", warp_idx);

        for (int i = 0; i < SharedStorage::kASizePerWarp; i += 8) {
          for (int j = 0; j < 8; ++j) {
            printf("%f, ", float(a_shared_ptr[i + j]));
          }
          printf("\n");
        }
      }
    }

    //
    // Mainloop
    //
    for (; proc_k < k_end; smem_write_stage = (smem_write_stage + 1) % kStages, smem_read_stage = (smem_read_stage + 1) % kStages, proc_k += WarpShape::kK){
      typename MetaLoader::FragmentScales fragment_addon;
  
      uint8_t* packed_b_smem_read_ptr = packed_b_shared_ptr + smem_read_stage * SharedStorage::kPackedBSizePerIter;
      uint8_t* packed_b_smem_write_ptr = packed_b_shared_ptr + smem_write_stage * SharedStorage::kPackedBSizePerIter;
      ElementT* a_smem_read_ptr = a_shared_ptr + smem_read_stage * SharedStorage::kASizePerIter;
      ElementT* a_smem_write_ptr = a_shared_ptr + smem_write_stage * SharedStorage::kASizePerIter;

      meta_loader.load(fragment_scales[smem_write_stage], load_k, min(k_end, load_k + WarpShape::kK));
      cutlass::Array<unsigned, PackedBLoader::kTiles>* packed_b_tile_frag_ptr =
          reinterpret_cast<cutlass::Array<unsigned, PackedBLoader::kTiles>*>(fragment_packed_b.data());

      meta_loader.process(fragment_scales[smem_read_stage], fragment_addon);

      // If PackedBLoader::kKStride > 16, then kNLoads must be 1. Because we don't want a
      // over-complicated tile visiting pattern. We always want to visit the all the
      // packed B tiles on the N dimension in a contiguous manner, and then move to the next
      // K dimension.
      static_assert(PackedBLoader::kKStride <= 16 || kB_Nloads == 1);

      // Load from shared memory to fragments/registers, and compute mma, 16 k at a time, dictated by Ampere mma shape
      CUTLASS_PRAGMA_UNROLL
      for (int warp_k_offset = 0; warp_k_offset < WarpShape::kK; warp_k_offset += InstructionShape::kK) {
        // Load packed weights. They are smaller in size, so they are loaded in bigger blocks
        if ((warp_k_offset % PackedBLoader::kKStride) == 0) {
          CUTLASS_PRAGMA_UNROLL
          for (int n_load = 0; n_load < kB_Nloads; ++n_load, ++packed_b_tile_frag_ptr) {
            PackedBLoader::ldmatrix_sync(*packed_b_tile_frag_ptr, lane_idx, packed_b_smem_read_ptr);
            packed_b_smem_read_ptr += PackedBLoader::kByteSize;

            if constexpr (kDebugPrintB) {
              uint8_t const* ptr = reinterpret_cast<uint8_t const*>(packed_b_tile_frag_ptr->data());
              printf("Warp: %d, lane %2d, smem_read_ptr %p, %3d %3d %3d %3d %3d %3d %3d %3d %3d %3d %3d %3d %3d %3d %3d %3d\n",
                warp_idx, lane_idx, packed_b_smem_read_ptr, ptr[0], ptr[1], ptr[2], ptr[3], ptr[4], ptr[5], ptr[6], ptr[7], ptr[8], ptr[9], ptr[10], ptr[11], ptr[12], ptr[13], ptr[14], ptr[15]);
            }
          }

          if (load_k < k_end) {
            packed_b_loader.load_lateral_n<kB_Nloads>(packed_b_smem_write_ptr);
            packed_b_smem_write_ptr += PackedBLoader::kByteSize * kB_Nloads;
            ++packed_b_loader;
          }
          load_k += PackedBLoader::kKStride;
        }

        static_assert(ATileLoader::kTiles * sizeof(unsigned) * kA_Mloads == FragmentA::kElements * sizeof(ElementT));
        cutlass::Array<unsigned, ATileLoader::kTiles>* a_tile_frag_ptr =
            reinterpret_cast<cutlass::Array<unsigned, ATileLoader::kTiles>*>(fragment_a.data());

        CUTLASS_PRAGMA_UNROLL
        for (int m_load = 0; m_load < kA_Mloads; ++m_load, ++a_tile_frag_ptr) {
          ATileLoader::ldmatrix_sync(*a_tile_frag_ptr, lane_idx, a_smem_read_ptr);
          a_smem_read_ptr += ATileLoader::kByteSize / kElementSize;
        }

        a_tile_loader.load_lateral_n<kA_Mloads>(a_smem_write_ptr);
        a_smem_write_ptr += ATileLoader::kByteSize * kA_Mloads / kElementSize;
        ++a_tile_loader;

        if constexpr (kDebugPrintFragA) {
          const int lane_id = threadIdx.x % 32;
          if (lane_id == 0) {
            printf("====  A tiles =======\n");
          }
          const char* const format = (lane_id == 31) ? "%f, %f\n\n" : ((lane_id % 4) == 3) ? "%f, %f\n" : "%f, %f, ";
          const ElementT* a_ptr = fragment_a.data();
          for (int m2_tile = 0; m2_tile < (WarpShape::kM / InstructionShape::kM); ++m2_tile, a_ptr += 8) {
            printf(format, float(a_ptr[0]), float(a_ptr[1]));
            printf(format, float(a_ptr[2]), float(a_ptr[3]));
            printf(format, float(a_ptr[4]), float(a_ptr[5]));
            printf(format, float(a_ptr[6]), float(a_ptr[7]));
          }
        }

        // Dequantize weights block (16, WarpShape::kN)
        meta_loader.dequant_k16(warp_k_offset, fragment_packed_b, fragment_scales[smem_read_stage], fragment_addon, fragment_b);

        // GEMM operation, covering a shape of (WarpShape::kM, WarpShape::kN, InstructionShape::kK)
        mma_op(accumulators, fragment_a, fragment_b, accumulators);
      }

      // Defines the boundary of a stage of cp.async.
      cutlass::arch::cp_async_fence();

      // Wait until we have at least one committed global fetch stage. (#uncommitted = Base::kStages - 1 - #committed)
      cutlass::arch::cp_async_wait<kStages - 2>();
      //__syncthreads(); is this necessary since the loader is warp based?

      if constexpr(kDebugPrintB) {
        if (lane_idx == 0) {
          printf("Mainloop, warp: %d, proc_k %d, load_k %d\nWritePtr: %p, ReadPtr: %p\n",
            warp_idx, proc_k, load_k, packed_b_smem_write_ptr, packed_b_smem_read_ptr);
        }
        cutlass::debug::dump_shmem(shared_storage.shared_B.data(), SharedStorage::kPackedBSize);
      }
    }

    if constexpr (kDebugPrintC) {
      static_assert(MmaOp::FragmentC::kElements == (WarpShape::kN / InstructionShape::kN) * (WarpShape::kM / InstructionShape::kM) * 4);
      for (int warp = 0; warp < kWarps; ++warp) {
        if (warp_idx == warp) {
          const float* c_ptr = accumulators.data();
          const int lane_id = threadIdx.x % 32;
          if (lane_id == 0) {
            printf("======= C tiles in warp %d =======\n", warp_idx);
          }
          const char* const format = (lane_id == 31) ? "%f, %f\n\n" : ((lane_id % 4) == 3) ? "%f, %f\n" : "%f, %f, ";
          for (int n_tile = 0; n_tile < (WarpShape::kN / InstructionShape::kN); ++n_tile) {
            for (int m_tile = 0; m_tile < (WarpShape::kM / InstructionShape::kM); ++m_tile, c_ptr += 4) {
              // since InstructionShape::kM is 16, we can print 2 tiles
              printf(format, float(c_ptr[0]), float(c_ptr[1]));
              printf(format, float(c_ptr[2]), float(c_ptr[3]));
            }
          }
        }
        __syncthreads();
      }
    }

    // Finished the main loop, now each warp (except warp 0) stores the partial results
    // to shared memory. Later warp 0 should gather them to form the final result
    using Float4 = cutlass::Array<float, 4>;  // hopefully utilize 128b st.shared.b128
    constexpr int kAccLoads = MmaOp::FragmentC::kElements / 4;
    static_assert(kAccLoads * 4 == MmaOp::FragmentC::kElements);
    if (warp_idx != 0){
      Float4* d_smem_ptr = reinterpret_cast<Float4*>(shared_storage.shared_Acc.data() + ((warp_idx - 1) * SharedStorage::kAccSizePerWarp));
      d_smem_ptr += lane_idx;
      Float4* f4s = reinterpret_cast<Float4*>(accumulators.data());
      CUTLASS_PRAGMA_UNROLL
      for (int acc_l = 0; acc_l < kAccLoads; ++acc_l) {
        d_smem_ptr[0] = f4s[acc_l];
        d_smem_ptr += 32;
      }
    }

    cutlass::arch::cp_async_wait<0>();
    if constexpr (kWarps > 1) {
      __syncthreads();
    }

    if (warp_idx != 0) {
      return;
    }

    //
    // Only warp 0 gathers the result from all other warps and stores it to global memory
    // Be extra careful with synchronization code below, as only a subset of threads
    // are active!
    //
    Float4 other_acc;

    CUTLASS_PRAGMA_UNROLL
    for (int warp = 1; warp < kWarps; ++warp) {
      Float4* d_smem_ptr = reinterpret_cast<Float4*>(shared_storage.shared_Acc.data() + ((warp - 1) * SharedStorage::kAccSizePerWarp));
      d_smem_ptr += lane_idx;

      if constexpr (kDebugPrintC) {
        if (lane_idx == 0) {
          printf("======= C gatered from warp %d =======\n", warp);
        }
      }

      CUTLASS_PRAGMA_UNROLL
      for (int acc_l = 0; acc_l < kAccLoads; ++acc_l) {
        other_acc = d_smem_ptr[0];
        d_smem_ptr += 32;

        if constexpr (kDebugPrintC) {
          const char* const format = (lane_idx == 31) ? "%f, %f\n\n" : ((lane_idx % 4) == 3) ? "%f, %f\n" : "%f, %f, ";
          printf(format, float(other_acc[0]), float(other_acc[1]));
          printf(format, float(other_acc[2]), float(other_acc[3]));
        }

        accumulators[acc_l * 4 + 0] += other_acc[0];
        accumulators[acc_l * 4 + 1] += other_acc[1];
        accumulators[acc_l * 4 + 2] += other_acc[2];
        accumulators[acc_l * 4 + 3] += other_acc[3];
      }


    }


    // Store the result
    __half2* output_ptr = reinterpret_cast<__half2*>(params.ptr_output_);
    int output_stride = params.output_byte_stride_ / sizeof(__half2);
    const float2* c_ptr = reinterpret_cast<float2 const*>(accumulators.data());

    int n = n_start + lane_b_k_offset * 2;
    CUTLASS_PRAGMA_UNROLL
    for (int n_tile = 0; n_tile < (WarpShape::kN / 8); ++n_tile, n += 8) {
      int m = m_start + lane_b_n_offset;
      CUTLASS_PRAGMA_UNROLL
      for (int m_tile = 0; m_tile < (WarpShape::kM / 8); ++m_tile, m += 8, ++c_ptr) {
        if (n < n_end && m < m_end) {
          *(output_ptr + m * output_stride + n/2) = __float22half2_rn(c_ptr[0]);
        }
      }
    }

  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

template <
  typename QuantBlocking_,              ///! Shape of the quantization block, either 1xb or bx1
  typename WarpShape_,                  ///! Warp-scoped matrix multiply-accumulate
  int SplitKSerial_ = 1,                ///! How many warps to split the K dimension in the same MxN block
  int Stages_ = 4                       ///! Stages of the pipelined mainloop
>
class LoadPackedBTest {
 public:
  using QuantBlocking = QuantBlocking_;
  using WarpShape = WarpShape_;
  static constexpr int kSplitK = SplitKSerial_;
  static constexpr int kStages = Stages_;

  using TestKernel = LoadPackedBTestKernel<QuantBlocking, false, WarpShape, kSplitK, kStages>;
  using Args = typename TestKernel::Params;

  cutlass::Status run(
    hipStream_t stream,
    cutlass::gemm::GemmCoord const & problem_size,
    void* ptr_output,
    int output_byte_stride,
    void const *ptr_a,
    int a_byte_stride,
    void const *ptr_packed_b,
    int b_byte_stride,
    void const *ptr_scales,
    int scales_byte_stride) {

    Args args(problem_size, ptr_output, output_byte_stride,
              ptr_a, a_byte_stride, ptr_packed_b, b_byte_stride,
              ptr_scales, scales_byte_stride);
    cutlass::Status status = TestKernel::can_implement(args);
    if (status != cutlass::Status::kSuccess) {
      return status;
    }

    dim3 grid(args.grid_tiled_shape_.m(), args.grid_tiled_shape_.n(), args.grid_tiled_shape_.k());
    dim3 block(TestKernel::kThreadCount, 1, 1);

    hipError_t result;

    int smem_size = int(sizeof(typename TestKernel::SharedStorage));

    if (smem_size >= (48 << 10)) {
      result = hipFuncSetAttribute(reinterpret_cast<const void*>(cutlass::Kernel<TestKernel>),
                                    hipFuncAttributeMaxDynamicSharedMemorySize,
                                    smem_size);

      if (result != hipSuccess) {
        std::cerr << "Failed to obtain maximum shared memory size " << smem_size << " for kernel: "
                  << hipGetErrorString(result) << "\n";
        return cutlass::Status::kErrorInternal;
      }
    }
   
    cutlass::Kernel<TestKernel><<<grid, block, smem_size, stream>>>(args);

    return cutlass::Status::kSuccess;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename LayoutT, typename ElementT>
void print_tiled_tensor(cutlass::HostTensor<ElementT, LayoutT>& t) {
  for (int row = 0; row < t.extent()[0]; ++row) {
    for (int col = 0; col < t.extent()[1]; ++col) {
      printf("%f, ", static_cast<float>(t.at({row, col})));
      if (col % 8 == 7) {
        printf(", ");
      }
    }
    printf("\n");
    if (row % 8 == 7) {
      printf("\n");
    }
  }
}


template <typename QuantBlocking, typename WarpShape, int kSplitK, int kStages>
void test_load_packed_b(int m, int n, int k) {
  std::cout << "Testing Blocking: " << QuantBlocking::kRow << "x" << QuantBlocking::kColumn 
            << " WarpShape: " << WarpShape::kM << "x" << WarpShape::kN << "x" << WarpShape::kK
            << ", kSplitK: " << kSplitK << ", kStages: " << kStages;
  std::cout << ", m: " << m << ", n: " << n << ", k: " << k << std::endl;

  using Test = LoadPackedBTest<QuantBlocking, WarpShape, kSplitK, kStages>;
  Test test;
  cutlass::gemm::GemmCoord problem_size(m, n, k);

  constexpr bool has_offsets = false;
  using QuantBaseT = onnxruntime::test::BlkQuantizationRef<QuantBlocking, has_offsets>;
  using LayoutQMeta = typename QuantBaseT::LayoutQMeta;

  // fill the tensor with reduced bits fp16 seems to be necessary to avoid rounding errors
  // during test. Need to investigate further why.
  cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor> tensor_a({m, k});
  cutlass::reference::host::TensorFillRandomUniform(tensor_a.host_view(), 174321, 1.5f, -1.125f, 6);

  cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor> tensor_b({k, n});
  cutlass::reference::host::TensorFillRandomUniform(tensor_b.host_view(), 193456, 1.75f, -1.25f, 8);
  cutlass::HostTensor<uint8_t, cutlass::layout::ColumnMajor> q4_weights;
  cutlass::HostTensor<cutlass::half_t, LayoutQMeta> scales;
  cutlass::HostTensor<uint8_t, LayoutQMeta> offsets;

  QuantBaseT::QuantizeFp16To4Bit(tensor_b, q4_weights, scales, offsets);
  QuantBaseT::Dequantize4BitToFp16(tensor_b, q4_weights, scales, offsets);
  QuantBaseT::QuantizeFp16To4Bit(tensor_b, q4_weights, scales, offsets);
  cutlass::reference::host::TensorFill(tensor_b.host_view(), cutlass::half_t(0));

  cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor> dst;
  QuantBaseT::Dequantize4BitToFp16(dst, q4_weights, scales, offsets);

  // Allocate result tensor
  cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor> tensor_d(
      problem_size.mn());
  cutlass::reference::host::TensorFill(tensor_d.host_view());

#if 0
  // Debug print the weights tensor detail
  for (int row = 0; row < k; ++row) {
    for (int col = 0; col < n; ++col) {
      auto weight_pos = cutlass::make_Coord(row/2, col);
      auto meta_pos = cutlass::make_Coord(row / QuantBlocking::kRow, col / QuantBlocking::kColumn);
      const float scale = static_cast<float>(scales.at(meta_pos));
      const uint8_t offset = has_offsets ? offsets.at(meta_pos) : 8;
      const int w = (row % 2 == 0) ? (q4_weights.at(weight_pos) & 0xf) : (q4_weights.at(weight_pos) >> 4);

      const float f = scale * (w - offset);
      printf("%f=%2dx%f,  ", float(dst.at({row, col})), w, scale);
      ASSERT_EQ(dst.at({row, col}), cutlass::half_t(f));
    }
    printf("\n");
  }

  // Debug print the tensor A
  for (int row = 0; row < tensor_a.extent()[0]; ++row) {
    for (int col = 0; col < tensor_a.extent()[1]; ++col) {
      printf("%f, ", float(tensor_a.at({row, col})));
    }
    printf("\n");
  }
#endif

  std::vector<uint8_t> packed_w_ref(k * n / 2);
  mickey::MatrixRef<uint8_t, cutlass::layout::ColumnMajor, true> tensor_packed_w_ref(
      packed_w_ref, cutlass::make_Coord(k, n / 2));
  onnxruntime::cuda::test::prepack_weights_ref(k, n, onnxruntime::test::make_ConstMatrixRef(q4_weights), tensor_packed_w_ref);

  int meta_tensor_stride = scales.stride(0);
  thrust::device_vector<cutlass::half_t> packed_scale_dev;

  if constexpr (std::is_same<LayoutQMeta, cutlass::layout::ColumnMajor>::value) {
    std::vector<cutlass::half_t> packed_scales_ref(scales.size());
    mickey::MatrixRef<cutlass::half_t, LayoutQMeta, true> tensor_packed_s_ref =
        mickey::make_MatrixRef<cutlass::half_t, LayoutQMeta, true>(packed_scales_ref, scales.extent());
    onnxruntime::cuda::test::prepack_quant_scales_ref<cutlass::half_t, LayoutQMeta, QuantBlocking>(
        k, n, onnxruntime::test::make_ConstMatrixRef(scales), tensor_packed_s_ref);
    packed_scale_dev = packed_scales_ref;
  
    // std::vector<uint8_t> packed_zp_ref(meta_shape.product());
    // mickey::MatrixRef<uint8_t, LayoutQMeta, true> tensor_packed_zp_ref =
    //     mickey::make_MatrixRef<ElementQOffset, LayoutQMeta, true>(packed_zp_ref, meta_shape);
    // onnxruntime::cuda::test::prepack_quant_offsets_ref<LayoutQMeta, QuantBlocking>(
    //       rows, columns, tensor_offset.const_ref(), tensor_packed_zp_ref);
  } else {
    packed_scale_dev.resize(scales.size());
    thrust::copy(scales.host_data(), scales.host_data() + scales.size(), packed_scale_dev.begin());
  }

  thrust::device_vector<uint8_t> packed_w_dev(packed_w_ref);
  tensor_d.sync_device();
  tensor_a.sync_device();

  cutlass::Status status = test.run(nullptr, problem_size,
                                    tensor_d.device_data(), tensor_d.stride(0) * sizeof(cutlass::half_t),
                                    tensor_a.device_data(), tensor_a.stride(0) * sizeof(cutlass::half_t),
                                    thrust::raw_pointer_cast(packed_w_dev.data()), problem_size.k(),
                                    thrust::raw_pointer_cast(packed_scale_dev.data()), meta_tensor_stride * sizeof(cutlass::half_t));
  ASSERT_EQ(status, cutlass::Status::kSuccess);
  tensor_d.sync_host();

  // Run reference kernel
  cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor> tensor_ref_d(
      problem_size.mn());  // <- Create matrix D with dimensions M x N used to store output from
                           // reference kernel
  cutlass::reference::host::TensorFill(tensor_ref_d.host_view());
  cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor> tensor_c(
      problem_size.mn());
  cutlass::reference::host::TensorFill(tensor_c.host_view());

  tensor_ref_d.sync_device();
  tensor_c.sync_device();
  dst.sync_device();

  // Initialize alpha and beta for dot product computation
  float alpha = 1.0f;
  float beta = 0.0f;

  compute_gemm_ref<cutlass::half_t, cutlass::layout::RowMajor,
                   cutlass::half_t, cutlass::layout::RowMajor,
                   cutlass::half_t, cutlass::layout::RowMajor,
                   float, float>(
      problem_size,
      alpha,
      tensor_a.device_ref(),
      dst.device_ref(),
      beta,
      tensor_c.device_ref(),
      tensor_ref_d.device_ref());

  // Wait for kernels to finish
  hipDeviceSynchronize();

  tensor_ref_d.sync_host();

  for (int row = 0; row < tensor_d.extent()[0]; ++row) {
    for (int col = 0; col < tensor_d.extent()[1]; ++col) {
      float expected = tensor_ref_d.at({row, col});
      float actual = tensor_d.at({row, col});
      if (expected == actual) {
        continue;
      }
      float diff = fabs(expected - actual);
      if (diff < 2e-7) {
        continue;
      }
      float diff_ratio = fabs(expected - actual) / max(fabs(expected), fabs(actual)); 
      if (diff_ratio > 3e-3) {
        std::cerr << "Mismatch found at (" << row << ", " << col << "): " << expected << " != " << actual << " ratio: " << diff_ratio << std::endl;
        ASSERT_TRUE(false);
      }
    }
  }
}

TEST(TensorCoreLoader, PackedBTest) {
  test_load_packed_b<cutlass::MatrixShape<1, 16>, cutlass::gemm::GemmShape<16, 64, 16>, 4, 2>(31, 128 + 32, 1024 + 16);
  test_load_packed_b<cutlass::MatrixShape<128, 1>, cutlass::gemm::GemmShape<16, 64, 16>, 8, 3>(67, 128 + 16, 4096 + 128);

  test_load_packed_b<cutlass::MatrixShape<128,1>, cutlass::gemm::GemmShape<16, 16, 64>, 2, 3>(65, 48, 1024 + 128);
  test_load_packed_b<cutlass::MatrixShape<1, 64>, cutlass::gemm::GemmShape<16, 16, 64>, 4, 4>(1, 128, 4096 + 16);

  test_load_packed_b<cutlass::MatrixShape<1, 16>, cutlass::gemm::GemmShape<32, 32, 32>, 1, 3>(35, 48, 32 * 4 + 16);
  test_load_packed_b<cutlass::MatrixShape<16, 1>, cutlass::gemm::GemmShape<32, 32, 32>, 1, 2>(35, 48, 32 * 3 + 16);
  test_load_packed_b<cutlass::MatrixShape<1, 128>, cutlass::gemm::GemmShape<16, 32, 32>, 8, 3>(70, 128, 4096 + 16);
  test_load_packed_b<cutlass::MatrixShape<64, 1>, cutlass::gemm::GemmShape<64, 32, 32>, 2, 2>(70, 48, 64 * 7);

  test_load_packed_b<cutlass::MatrixShape<1, 32>, cutlass::gemm::GemmShape<64, 64, 128>, 1, 4>(68, 160, 4096 + 16);
  test_load_packed_b<cutlass::MatrixShape<32, 1>, cutlass::gemm::GemmShape<128, 128, 128>, 1, 4>(170, 176, 2048 + 32);
}

} // namespace test
} // namespace cuda
} // namespace onnxruntime
