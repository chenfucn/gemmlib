
#include "gemmlib.h"
#include "blk_q4/f16_gemm_sm80.h"

namespace mickey {

/**
 * @brief Helper function to run the GEMM kernel for 4bits quantized gemm on SM80.
 * Only support fp16 for now.
*/
template<
    int block_size,
    bool column_wise_blocking,
    bool small_m,
    bool has_offsets>
std::string blkq4_gemm_sm80(int m, int n, int k, hipStream_t stream,
                     gsl::span<half const> a,
                     gsl::span<uint8_t const> weights,
                     gsl::span<half const> scales,
                     gsl::span<uint8_t const> offsets,
                     gsl::span<half> output) {

  using ElementDequant = cutlass::half_t;
  using QuantBlocking =
    typename std::conditional<column_wise_blocking,
                     cutlass::MatrixShape<block_size, 1>,
                     cutlass::MatrixShape<1, block_size>>::type;

  using GemmRunner = BlkQ4F16GemmImpl<ElementDequant, QuantBlocking, small_m, has_offsets>;

  using ElementAccumulator = typename GemmRunner::ElementAccumulator;
  using ElementComputeEpilogue = typename GemmRunner::ElementComputeEpilogue;
  using ElementOutput = typename GemmRunner::ElementOutput;
  using ElementW = typename GemmRunner::ElementW;
  using ElementWPack = typename GemmRunner::ElementWPack;
  using ElementQScale = typename GemmRunner::ElementQScale;
  using ElementQOffset = typename GemmRunner::ElementQOffset;

  using LayoutInputA = typename GemmRunner::LayoutInputA;
  using LayoutOutput = typename GemmRunner::LayoutOutput;
  using LayoutInputWPack = typename GemmRunner::LayoutInputWPack;
  using LayoutInputQScale = typename GemmRunner::LayoutInputQScale;

  const cutlass::gemm::GemmCoord problem_size = {m, n, k};

  if (m == 0 || n == 0 || k == 0) {
    return std::string();
  }

  if (a.size_bytes() != m * k * sizeof(ElementDequant)) {
    return "Unexpected activation tensor size: " + std::to_string(a.size_bytes())
           + " expected: " + std::to_string(m * k * sizeof(ElementDequant));
  }
  cutlass::TensorRef<ElementDequant const, LayoutInputA> ref_a(
    reinterpret_cast<ElementDequant const *>(a.data()),
    LayoutInputA::packed({m, k}));

  if (weights.size_bytes() != (k/2) * (n/2) * sizeof(ElementWPack)) {
    return "Unexpected weight tensor size: " + std::to_string(weights.size_bytes())
           + " expected: " + std::to_string(k/2 * n/2 * sizeof(ElementWPack));
  }
  cutlass::TensorRef<ElementWPack const, LayoutInputWPack> ref_W(
    reinterpret_cast<ElementWPack const *>(weights.data()),
    LayoutInputWPack::packed({k/2, n/2}));

  if (scales.size_bytes() != (k/QuantBlocking::kRow) * (n/QuantBlocking::kColumn) * sizeof(ElementQScale)) {
    return "Unexpected scale tensor size: " + std::to_string(scales.size_bytes())
           + " expected: " + std::to_string((k/QuantBlocking::kRow) * (n/QuantBlocking::kColumn) * sizeof(ElementQScale));
  }
  cutlass::TensorRef<ElementQScale const, LayoutInputQScale> ref_scales(
    reinterpret_cast<ElementQScale const *>(scales.data()),
    LayoutInputQScale::packed({k/QuantBlocking::kRow, n/QuantBlocking::kColumn}));

  if (output.size_bytes() != m * n * sizeof(ElementOutput)) {
    return "Unexpected output tensor size: " + std::to_string(output.size_bytes())
           + " expected: " + std::to_string(m * n * sizeof(ElementOutput));
  }

  cutlass::TensorRef<ElementOutput, LayoutOutput> ref_output(
    reinterpret_cast<ElementOutput *>(output.data()),
    LayoutOutput::packed({m, n}));

  // run GEMM
  cutlass::Status status;
  if constexpr (has_offsets) {
    if (offsets.size_bytes() != (k/QuantBlocking::kRow) * (n/QuantBlocking::kColumn) * sizeof(ElementQOffset)) {
      return "Unexpected offsets tensor size: " + std::to_string(offsets.size_bytes())
             + " expected: " + std::to_string((k/QuantBlocking::kRow) * (n/QuantBlocking::kColumn) * sizeof(ElementQOffset));
    }
    cutlass::TensorRef<ElementQOffset const, LayoutInputQScale> ref_offsets(
      reinterpret_cast<ElementQOffset const *>(offsets.data()),
      LayoutInputQScale::packed({k/QuantBlocking::kRow, n/QuantBlocking::kColumn}));
    status = GemmRunner::run(
      stream, problem_size, ref_a, ref_W, ref_scales, ref_offsets,
      ref_output, ref_output);
  } else {
    status = GemmRunner::run(
      stream, problem_size, ref_a, ref_W, ref_scales,
      ref_output, ref_output);
  }
  if (status != cutlass::Status::kSuccess) {
    return "Kernel execution failed: " + std::string(cutlassGetStatusString(status));
  }
  return std::string();
}

std::string blkq4_fp16_gemm_sm80_dispatch(
  int block_size,
  bool column_wise_blocking,
  int m, int n, int k, hipStream_t stream,
  gsl::span<half const> a,
  gsl::span<uint8_t const> weights,
  gsl::span<half const> scales,
  gsl::span<uint8_t const> offsets,
  gsl::span<half> output) {

  switch (block_size)
  {
  case 16:
    if (column_wise_blocking) {
      if (m > 16) {
        if (offsets.empty())
          return blkq4_gemm_sm80<16, true, false, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<16, true, false, true>(m, n, k, stream, a, weights, scales, offsets, output);
      } else {
        if (offsets.empty())
          return blkq4_gemm_sm80<16, true, true, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<16, true, true, true>(m, n, k, stream, a, weights, scales, offsets, output);
      }
    } else {
      if (m > 16) {
        if (offsets.empty())
          return blkq4_gemm_sm80<16, false, false, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<16, false, false, true>(m, n, k, stream, a, weights, scales, offsets, output);
      } else {
        if (offsets.empty())
          return blkq4_gemm_sm80<16, false, true, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<16, false, true, true>(m, n, k, stream, a, weights, scales, offsets, output);
      }
    }
    break;

  case 32:
    if (column_wise_blocking) {
      if (m > 16) {
        if (offsets.empty())
          return blkq4_gemm_sm80<32, true, false, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<32, true, false, true>(m, n, k, stream, a, weights, scales, offsets, output);
      } else {
        if (offsets.empty())
          return blkq4_gemm_sm80<32, true, true, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<32, true, true, true>(m, n, k, stream, a, weights, scales, offsets, output);
      }
    } else {
      if (m > 16) {
        if (offsets.empty())
          return blkq4_gemm_sm80<32, false, false, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<32, false, false, true>(m, n, k, stream, a, weights, scales, offsets, output);
      } else {
        if (offsets.empty())
          return blkq4_gemm_sm80<32, false, true, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<32, false, true, true>(m, n, k, stream, a, weights, scales, offsets, output);
      }
    }
    break;

  case 64:
    if (column_wise_blocking) {
      if (m > 16) {
        if (offsets.empty())
          return blkq4_gemm_sm80<64, true, false, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<64, true, false, true>(m, n, k, stream, a, weights, scales, offsets, output);
      } else {
        if (offsets.empty())
          return blkq4_gemm_sm80<64, true, true, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<64, true, true, true>(m, n, k, stream, a, weights, scales, offsets, output);
      }
    } else {
      if (m > 16) {
        if (offsets.empty())
          return blkq4_gemm_sm80<64, false, false, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<64, false, false, true>(m, n, k, stream, a, weights, scales, offsets, output);
      } else {
        if (offsets.empty())
          return blkq4_gemm_sm80<64, false, true, false>(m, n, k, stream, a, weights, scales, offsets, output);
        else
          return blkq4_gemm_sm80<64, false, true, true>(m, n, k, stream, a, weights, scales, offsets, output);
      }
    }
    break;
  }

  return "Unsupported block size: " + std::to_string(block_size);
}

}  // namespace mickey
